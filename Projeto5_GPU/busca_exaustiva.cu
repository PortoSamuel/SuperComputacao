#include "hip/hip_runtime.h"

#include <chrono>
#include <vector>
#include <iostream>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using namespace std;

struct nucleotidio
{
    int id;
    char base;
};

struct mat_ele
{
    int i = -1, j = -1, val = 0;
};

struct result
{
    int score = 0;
    vector<nucleotidio> subseq_a;
    vector<nucleotidio> subseq_b;
};

// Function to create all the power set
vector<vector<nucleotidio>> createPowerSet(vector<nucleotidio> set, int set_size)
{
    vector<vector<nucleotidio>> power_set_list;
    vector<nucleotidio> subseq;

    // Set_size of power set of a set with set_size
    // n is (2^n-1)
    int pow_set_size = pow(2, set_size);
    int counter, j;

    // Run from counter 000..0 to 111..1
    for (counter = 0; counter < pow_set_size; counter++)
    {
        for (j = 0; j < set_size; j++)
        {
            // Check if jth bit in the counter is set
            // If set then save jth element from set
            if (counter & (1 << j))
            {
                nucleotidio n;
                n.id = j;
                n.base = set[j].base;
                subseq.push_back(n);
            }
        }

        if (subseq.size() > 0)
        {
            // verify if the elements in the subsequence are subseq of the original set
            bool is_subseq = true;
            int size = subseq.size();

            for (int i = 0; i < size - 1; i++)
            {
                if (subseq[i + 1].id == subseq[i].id + 1)
                {
                    is_subseq = true;
                }
                else
                {
                    is_subseq = false;
                    break;
                }
            }

            if (is_subseq)
            {
                power_set_list.push_back(subseq);
            }

            subseq.clear();
        }
    }

    return power_set_list;
}

int char_score(char a, char b)
{
    if (a == b && b != '-')
    {
        return 2;
    }
    else
    {
        return -1;
    }
}

// Função que calcula o score de duas subsequencias de mesmo tamanho
// Como feito no projeto 2
// int w(vector<nucleotidio> sa, vector<nucleotidio> sb)
// {
//     int score = 0;
//     int u = sa.size();

//     for (int i = 0; i < u; i++)
//     {
//         if (sa[i].base == sb[i].base && sb[i].base != '-')
//         {
//             score += 2;
//         }
//         else
//         {
//             score -= 1;
//         }
//     }

//     return score;
// }

// Função que calcula o score de duas subsequencias de mesmo tamanho paralelizando na GPU
template <typename T>
struct w
{
    int value;
	  typedef int result_type;
	  typedef T first_argument_type;
	  typedef T second_argument_type;
  
    __host__ __device__
        T operator()(const T &x, const T &y) const { 
            value = 0;

              if (x.base == y.base && y.base != '-')
              {
                  value += 2;
              }
              else
              {
                  value -= 1;
              }

            return value;
        }
};

// Algoritmo de smith-waterman para calcular o score de subsequencias com diferentes tamanhos
// int Smith_Waterman(vector<nucleotidio> seq_a, vector<nucleotidio> seq_b)
// {
//     int n = seq_a.size();
//     int m = seq_b.size();

//     vector<vector<mat_ele>> H(n + 1, vector<mat_ele>(m + 1));

//     for (int i = 0; i < n + 1; i++)
//     {
//         for (int j = 0; j < m + 1; j++)
//         {
//             mat_ele item;

//             if (j == 0 || i == 0)
//             {
//                 H[i][j] = item;
//             }
//             else
//             {

//                 int diagonal = H[i - 1][j - 1].val + char_score(seq_a[i - 1].base, seq_b[j - 1].base);
//                 int delecao = H[i - 1][j].val - 1;
//                 int insercao = H[i][j - 1].val - 1;

//                 if (max({0, diagonal, delecao, insercao}) == diagonal)
//                 {
//                     item.i = i - 1;
//                     item.j = j - 1;
//                 }
//                 else if (max({0, diagonal, delecao, insercao}) == delecao)
//                 {
//                     item.i = i - 1;
//                     item.j = j;
//                 }
//                 else if (max({0, diagonal, delecao, insercao}) == insercao)
//                 {
//                     item.i = i;
//                     item.j = j - 1;
//                 }

//                 item.val = max({0, diagonal, delecao, insercao});

//                 H[i][j] = item;
//             }
//         }
//     }

//     mat_ele max;

//     for (auto &el1 : H)
//     {
//         for (auto &el2 : el1)
//         {
//             if (el2.val > max.val)
//             {
//                 max = el2;
//             }
//         }
//     }

//     return max.val;
// }

int main()
{
    int n = 0;
    int m = 0;

    srand(24); // "Random" Seed

    cin >> n >> m;

    nucleotidio element;
    result resultado;

    // Sequencias A e B
    thrust::host_vector<nucleotidio> a(n);
    thrust::host_vector<nucleotidio> b(m);

    // Listas com todos os subconjuntos de A e B
    thrust::host_vector<thrust::host_vector<nucleotidio>> power_set_a;
    thrust::host_vector<thrust::host_vector<nucleotidio>> power_set_b;

    // Captura os elementos da primeira sequencia
    for (int i = 0; i < n; i++)
    {
        element.id = i;
        cin >> element.base;

        a[i] = element;
    }

    // Captura os elementos da segunda sequencia
    for (int i = 0; i < m; i++)
    {
        element.id = i;
        cin >> element.base;

        b[i] = element;
    }

    // Garantindo que a sequencia a seja a maior que a b
    // Para que as subsequencias de tamanho k não sejam maior que a sequencia a
    if (m > n)
    {
        int aux = n;
        vector<nucleotidio> temp = a;

        a = b;
        b = temp;

        n = m;
        m = aux;
    }
 
    // Enviando para a GPU as sequencias A e B
    thrust::device_vector<nucleotidio> gpu_a(a);
	  thrust::device_vector<nucleotidio> gpu_b(b);

    // Gera todos os subconjuntos de A e B e armazena em powerset_a e powerset_b
    powerset_a = createPowerSet(a, n);
    powerset_b = createPowerSet(b, m);
 
    // Enviando para a GPU os powersets de A e B
    thrust::device_vector<thrust::device_vector<nucleotidio>> gpu_powerset_a(powerset_a);
	  thrust::device_vector<thrust::device_vector<nucleotidio>> gpu_powerset_b(powerset_b);

    // Calcula o score de todas as subsequencias de A e B
    for (auto &el : gpu_powerset_a)
    {
        for (auto &el2 : gpu_powerset_b)
        {
            int temp_score = 0;

            if (el2.size() == el.size())
            {
                thrust::device_vector<nucleotidio> res(el.size());
                thrust::transform(el.begin(), el.end(), el2.end(), res.begin(), w());
             
                temp_score = thrust::reduce(res.begin(), res.end(), 0);

            }
            else
            {
                // temp_score = Smith_Waterman(el, el2);
            }

            // Verifica se o temp_score é maior que o score do resultado
            if (temp_score > resultado.score)
            {
                resultado.score = temp_score;
                resultado.subseq_a = el;
                resultado.subseq_b = el2;
            }
        }
    }

    // Imprime o resultado e as subsequencias
    cout << "subsequencia A: ";
    for (auto &el : resultado.subseq_a)
    {
        cout << el.base;
    }
    cout << endl;

    cout << "subsequencia B: ";
    for (auto &el : resultado.subseq_b)
    {
        cout << el.base;
    }
    cout << endl;

    cout << "Score: " << resultado.score << endl;

    return 0;
}
